
#include <hip/hip_runtime.h>
#include <iostream>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void add(float* A, float* B, float* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        C[tid] = A[tid] + B[tid];
}

void maxError(float *output, int iterations, int N, bool isC) {
    float maxErr = 0;
    int maxI = -1;
    const float epsilon = 1e-6;
    for (int i = 0; i < N; ++i) {
        float expected = isC ? 3.0f : 6.0f;
        float diff = std::abs(output[i] - expected);
        if (diff > maxErr) {
            maxErr = diff;
            maxI = i;
        }
        if (diff > epsilon) {
            std::cerr << "Error: " << (isC ? "C" : "F") << ": " << i << " (" << output[i] << ")!" << std::endl;
        }
    }
    std::cout << "Max error: " << maxErr << std::endl;
    if (maxI != -1) {
        std::cout << "at " << maxI << ": " << output[maxI] << std::endl;
    }
}

float experiment(bool overlap, int N, int iterations) {
    size_t size = N * sizeof(float);
    int block = 256;
    int grid = (N + block - 1) / block;

    float *d_A, *d_B, *d_C;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipMalloc((void**)&d_A, size));
    checkCudaError(hipMalloc((void**)&d_B, size));
    checkCudaError(hipMalloc((void**)&d_C, size));

    float *d_D, *d_E, *d_F;
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMalloc((void**)&d_D, size));
    checkCudaError(hipMalloc((void**)&d_E, size));
    checkCudaError(hipMalloc((void**)&d_F, size));

    float *h_A, *h_B, *h_C, *h_D, *h_F;
    checkCudaError(hipHostMalloc((void**)&h_A, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_B, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_C, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_D, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_F, size, hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
        h_C[i] = -100.0f;
        h_D[i] = 3.0f;
    }

    hipEvent_t startEvent, endEvent, copyEvent;
    checkCudaError(hipEventCreate(&startEvent));
    checkCudaError(hipEventCreate(&endEvent));

    hipStream_t stream0, stream1;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipStreamCreate(&stream0));
    checkCudaError(hipEventCreate(&copyEvent));
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipStreamCreate(&stream1));

    checkCudaError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice));

    checkCudaError(hipEventRecord(startEvent, 0));

    for (int i = 0; i < iterations; ++i) {
        checkCudaError(hipSetDevice(0));
        // puts("first add:");
        add<<<grid, block, 0, stream0>>>(d_A, d_B, d_C, N);
        checkCudaError(hipGetLastError());
        // puts("first add finished");
        if (overlap) {
            // checkCudaError(cudaMemcpyPeerAsync(d_E, 1, d_C, 0, size, stream0));
            // puts("overlap copy:");
            checkCudaError(hipMemcpyAsync(d_E, d_C, size, hipMemcpyDeviceToDevice, stream0));
            // puts("overlap copy launched");
            checkCudaError(hipEventRecord(copyEvent, stream0));
        } else {
            // puts("sync copy:");
            // checkCudaError(cudaStreamSynchronize(stream0));
            // checkCudaError(cudaMemcpyPeer(d_E, 1, d_C, 0, size));
            // checkCudaError(cudaMemcpy(d_E, d_C, size, cudaMemcpyDeviceToDevice));
            checkCudaError(hipMemcpyAsync(d_E, d_C, size, hipMemcpyDeviceToDevice, stream0));
            // puts("sync copy launched");
            // puts("syncing stream0:");
            checkCudaError(hipStreamSynchronize(stream0));
            // puts("synced");
            // checkCudaError(cudaEventRecord(copyEvent, stream0));
            // checkCudaError(cudaStreamSynchronize(0));
            // puts("recording copy event:");
            checkCudaError(hipEventRecord(copyEvent, stream0));
            // puts("recorded");
        }
        checkCudaError(hipSetDevice(1));
        checkCudaError(hipStreamWaitEvent(stream1, copyEvent, 0));
        // checkCudaError(cudaEventSynchronize(copyEvent));

        // puts("second add:");
        add<<<grid, block, 0, stream1>>>(d_D, d_E, d_F, N);
        checkCudaError(hipGetLastError());
    }

    checkCudaError(hipStreamSynchronize(stream0));
    checkCudaError(hipStreamSynchronize(stream1));
    checkCudaError(hipEventRecord(endEvent, 0));
    checkCudaError(hipEventSynchronize(endEvent));
    // checkCudaError(cudaSetDevice(0));
    // checkCudaError(cudaDeviceSynchronize());
    // checkCudaError(cudaSetDevice(1));
    // checkCudaError(cudaDeviceSynchronize());

    float elapse;
    checkCudaError(hipEventElapsedTime(&elapse, startEvent, endEvent));

    std::cout << "Total time (" << (overlap ? "Overlapping" : "Synchronous") << "): " << elapse << " ms\n";

    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_F, d_F, size, hipMemcpyDeviceToHost));
    // checkCudaError(cudaMemcpyAsync(h_C, d_C, size, cudaMemcpyDeviceToHost, stream0));
    // checkCudaError(cudaMemcpyAsync(h_F, d_F, size, cudaMemcpyDeviceToHost, stream1));
    checkCudaError(hipStreamSynchronize(stream0));
    checkCudaError(hipStreamSynchronize(stream1));

    maxError(h_C, iterations, N, true);
    maxError(h_F, iterations, N, false);

    checkCudaError(hipSetDevice(0));
    checkCudaError(hipFree(d_A));
    checkCudaError(hipFree(d_B));
    checkCudaError(hipFree(d_C));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipFree(d_D));
    checkCudaError(hipFree(d_E));
    checkCudaError(hipFree(d_F));

    checkCudaError(hipEventDestroy(startEvent));
    checkCudaError(hipEventDestroy(endEvent));
    checkCudaError(hipEventDestroy(copyEvent));

    checkCudaError(hipStreamDestroy(stream0));
    checkCudaError(hipStreamDestroy(stream1));

    checkCudaError(hipHostFree(h_A));
    checkCudaError(hipHostFree(h_B));
    checkCudaError(hipHostFree(h_C));
    checkCudaError(hipHostFree(h_D));
    checkCudaError(hipHostFree(h_F));

    return elapse;
}

int main() {
    const int experiments = 1;
    const int Ns[experiments] = {10};
    // const int experiments = 11;
    // for N in [417, 1499], CUBLAS_STATUS_EXECUTION_FAILED error???
    // const int Ns[experiments] = {400, 1500, 2000, 3000, 4000, 5000, 6000, 7000, 8000, 9000, 10000};
    // const int Ns[experiments] = {400, 11000, 12000, 13000, 14000, 15000, 16000, 17000, 18000, 19000, 20000};
    const int iterations = 10;

    for (int i = 0; i < experiments; i++) {
        const int N = Ns[i];
        std::cout << "=============================="
                << std::endl << "N = " << N << std::endl;
        std::cout << "Running synchronous version...\n";
        float sync = experiment(false, N, iterations);

        std::cout << "Running overlapping version...\n";
        float async = experiment(true, N, iterations);

        float speedup = (sync - async) / (sync);
        std::cout << "speedup: " << speedup << std::endl;
    }

    return 0;
}
