
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << __LINE__ << std::endl;
        exit(EXIT_FAILURE);
    }
}

inline void checkCublasError(hipblasStatus_t err) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error: " << err << " at line " << __LINE__ << std::endl;
        exit(1);
    }
}

void maxError(float *output, int iterations, int N, bool isC) {
    float maxErr = 0;
    int maxI = -1, maxJ = -1;
    const float epsilon = 1;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < i; ++j) {
            float diff = std::abs(output[i * N + j]);
            if (diff > maxErr) {
                maxErr = diff;
                maxI = i;
                maxJ = j;
            }
            if (diff > epsilon) {
                std::cerr << "Error: " << (isC ? "C" : "F") << ": " << i << ", " << j << " (" << output[i * N + j] << ")!" << std::endl;
            }
        }
        {
            float diff = std::abs(std::pow((float) i / N, isC ? iterations : iterations + 2) - output[i * N + i]);
            if (diff > maxErr) {
                maxErr = diff;
                maxI = i;
                maxJ = i;
            }
            if (diff > epsilon) {
                std::cerr << "Error: " << (isC ? "C" : "F") << ": " << i << ", " << i << " (" << output[i * N + i] << ")!" << std::endl;
            }
        }
        for (int j = i + 1; j < N; ++j) {
            float diff = std::abs(output[i * N + j]);
            if (diff > maxErr) {
                maxErr = diff;
                maxI = i;
                maxJ = j;
            }
            if (diff > epsilon) {
                std::cerr << "Error: " << (isC ? "C" : "F") << ": " << i << ", " << j << " (" << output[i * N + j] << ")!" << std::endl;
            }
        }
    }
    std::cout << "Max error: " << maxErr << std::endl;
    if (maxI != -1) {
        std::cout << "at " << maxI << ", " << maxJ << ": " << output[maxI * N + maxJ] << std::endl;
    }
}

float experiment(bool overlap, int N, int iterations) {
    if (iterations % 2 != 0) {
        std::cerr << "#iterations must be even" << std::endl;
        iterations--;
    }
    size_t size = N * N * sizeof(float);
    const float alpha = 1.0f;
    const float beta = 0.0f;

    float *d_A, *d_B, *d_C;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipMalloc((void**)&d_A, size));
    checkCudaError(hipMalloc((void**)&d_B, size));
    checkCudaError(hipMalloc((void**)&d_C, size));

    float *d_D, *d_E, *d_F;
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMalloc((void**)&d_D, size));
    checkCudaError(hipMalloc((void**)&d_E, size));
    checkCudaError(hipMalloc((void**)&d_F, size));

    float *h, *h_C, *h_F;
    // h = (float*)malloc(size);
    // h_C = (float*)malloc(size);
    // h_F = (float*)malloc(size);
    checkCudaError(hipHostMalloc((void**)&h, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_C, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_F, size, hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (j == i) {
                h[i * N + i] = (float) i / N;
            } else {
                h[i * N + j] = 0;
            }
        }
    }

    hipblasHandle_t handle0, handle1;
    checkCublasError(hipblasCreate(&handle0));
    checkCublasError(hipblasCreate(&handle1));

    float elapse;

    hipEvent_t startEvent, endEvent;
    checkCudaError(hipEventCreate(&startEvent));
    checkCudaError(hipEventCreate(&endEvent));

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream));

    checkCudaError(hipMemcpy(d_A, h, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_B, h, size, hipMemcpyHostToDevice));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMemcpy(d_D, h, size, hipMemcpyHostToDevice));

    checkCudaError(hipEventRecord(startEvent, 0));

    for (int i = 0; i < iterations / 2; ++i) {
        checkCudaError(hipSetDevice(0));
        checkCublasError(hipblasSgemm(handle0, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));
        // heavier computation leads to higher speedup
        // checkCublasError(cublasSgemm(handle0, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));
        // checkCublasError(cublasSgemm(handle0, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));
        if (overlap) {
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_C, 0, size, stream));
        } else {
            checkCudaError(hipMemcpyPeer(d_E, 1, d_C, 0, size));
            // checkCudaError(cudaStreamSynchronize(0));
        }
        checkCudaError(hipSetDevice(1));
        checkCublasError(hipblasSgemm(handle1, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_D, N, d_E, N, &beta, d_F, N));

        checkCudaError(hipSetDevice(0));
        checkCublasError(hipblasSgemm(handle0, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_C, N, d_B, N, &beta, d_A, N));
        if (overlap) {
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_A, 0, size, stream));
        } else {
            checkCudaError(hipMemcpyPeer(d_E, 1, d_A, 0, size));
        }
        checkCudaError(hipSetDevice(1));
        checkCublasError(hipblasSgemm(handle1, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_D, N, d_E, N, &beta, d_F, N));
    }

    // checkCudaError(cudaSetDevice(0));
    // checkCudaError(cudaDeviceSynchronize());  // Wait for all operations to complete
    // checkCudaError(cudaSetDevice(1));
    // checkCudaError(cudaDeviceSynchronize());  // Wait for all operations to complete
    checkCudaError(hipEventRecord(endEvent, 0));
    checkCudaError(hipEventSynchronize(endEvent));
    checkCudaError(hipEventElapsedTime(&elapse, startEvent, endEvent));

    std::cout << "Total time (" << (overlap ? "Overlapping" : "Synchronous") << "): " << elapse << " ms\n";

    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_F, d_F, size, hipMemcpyDeviceToHost));

    maxError(h_C, iterations, N, true);
    maxError(h_F, iterations, N, false);

    checkCudaError(hipSetDevice(0));
    checkCudaError(hipFree(d_A));
    checkCudaError(hipFree(d_B));
    checkCudaError(hipFree(d_C));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipFree(d_D));
    checkCudaError(hipFree(d_E));
    checkCudaError(hipFree(d_F));

    checkCudaError(hipEventDestroy(startEvent));
    checkCudaError(hipEventDestroy(endEvent));

    checkCudaError(hipStreamDestroy(stream));

    checkCudaError(hipHostFree(h));
    checkCudaError(hipHostFree(h_C));
    checkCudaError(hipHostFree(h_F));
    // free(h);
    // free(h_C);
    // free(h_F);

    return elapse;
}

int main() {
    const int experiments = 11;
    // for N in [417, 1499], CUBLAS_STATUS_EXECUTION_FAILED error???
    const int Ns[experiments] = {400, 1500, 2000, 3000, 4000, 5000, 6000, 7000, 8000, 9000, 10000};
    // const int Ns[experiments] = {400, 11000, 12000, 13000, 14000, 15000, 16000, 17000, 18000, 19000, 20000};
    const int iterations = 10;

    for (int i = 0; i < experiments; i++) {
        const int N = Ns[i];
        std::cout << "=============================="
                << std::endl << "N = " << N << std::endl;
        std::cout << "Running synchronous version...\n";
        float sync = experiment(false, N, iterations);

        std::cout << "Running overlapping version...\n";
        float async = experiment(true, N, iterations);

        float speedup = (sync - async) / (sync);
        std::cout << "speedup: " << speedup << std::endl;
    }

    return 0;
}
