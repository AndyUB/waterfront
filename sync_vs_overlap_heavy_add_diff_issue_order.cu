
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void add(float* A, float* B, float* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        for (int i = 0; i < 32; i++) {
            float sinA = sinf(A[tid]);
            float cosA = cosf(A[tid]);
            float idA = sqrtf(sinA * sinA + cosA * cosA) * A[tid];
            float sinB = sinf(B[tid]);
            float cosB = cosf(B[tid]);
            float idB = sqrtf(sinB * sinB + cosB * cosB) * B[tid];
            float added = idA + idB;
            float sinAdded = sinf(added);
            float cosAdded = cosf(added);
            float idAdded = sqrtf(sinAdded * sinAdded + cosAdded * cosAdded) * added;
            C[tid] = idAdded;
        }
    }
}

void maxError(float *output, int iterations, int N, bool isC) {
    float maxErr = 0;
    int maxI = -1;
    const float epsilon = 1e-6;
    // basic comp:
    float expected = isC ? 3.0f : 6.0f;
    // heavy comp:
    // float expected = 4 * iterations + 3;
    // heavier comp:
    // float expected = 8 * iterations + 3;
    // if (!isC) expected += 3;
    for (int i = 0; i < N; ++i) {
        float diff = std::abs(output[i] - expected);
        if (diff > maxErr) {
            maxErr = diff;
            maxI = i;
        }
        if (diff > epsilon) {
            std::cout << "Error: " << (isC ? "C" : "F") << ": " << i << " (" << output[i] << ")!" << std::endl;
        }
    }
    std::cout << "Max error: " << maxErr << std::endl;
    if (maxI != -1) {
        std::cout << "at " << maxI << ": " << output[maxI] << std::endl;
    }
}

float experiment(bool overlap, int N, int iterations) {
    // if (iterations % 2 != 0) {
    //     std::cout << "iterations must be even" << std::endl;
    //     iterations--;
    // }
    size_t size = N * sizeof(float);
    int block = 256;
    int grid = (N + block - 1) / block;

    float *d_A, *d_B, *d_C, *d_X;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipMalloc((void**)&d_A, size));
    checkCudaError(hipMalloc((void**)&d_B, size));
    checkCudaError(hipMalloc((void**)&d_C, size));
    checkCudaError(hipMalloc((void**)&d_X, size));

    float *d_D, *d_E, *d_F, *d_Y;
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMalloc((void**)&d_D, size));
    checkCudaError(hipMalloc((void**)&d_E, size));
    checkCudaError(hipMalloc((void**)&d_F, size));
    checkCudaError(hipMalloc((void**)&d_Y, size));

    float *h_A, *h_B, *h_C, *h_D, *h_F;
    checkCudaError(hipHostMalloc((void**)&h_A, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_B, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_C, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_D, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_F, size, hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
        h_C[i] = -100.0f;
        h_D[i] = 3.0f;
    }

    hipEvent_t startEvent, endEvent,
            copyCEvent, copyXEvent,
            compCEvent, compXEvent,
            compFEvent, compYEvent;
    checkCudaError(hipEventCreate(&startEvent));
    checkCudaError(hipEventCreate(&endEvent));

    hipStream_t compute0, compute1, copyStream;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipStreamCreate(&compute0));
    checkCudaError(hipEventCreate(&compCEvent));
    checkCudaError(hipEventCreate(&compXEvent));
    checkCudaError(hipStreamCreate(&copyStream));
    checkCudaError(hipEventCreate(&copyCEvent));
    checkCudaError(hipEventCreate(&copyXEvent));
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipStreamCreate(&compute1));
    checkCudaError(hipEventCreate(&compFEvent));
    checkCudaError(hipEventCreate(&compYEvent));

    checkCudaError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice));

    checkCudaError(hipEventRecord(startEvent, 0));
    const auto chronoStart = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < iterations; ++i) {
        checkCudaError(hipSetDevice(0));

        checkCudaError(hipStreamWaitEvent(compute0, copyCEvent, 0));
        add<<<grid, block, 0, compute0>>>(d_A, d_B, d_C, N);  // C_1 = A_0 + B_0
        checkCudaError(hipGetLastError());

        if (overlap) {
            checkCudaError(hipEventRecord(compCEvent, compute0));
        } else {
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_C, 0, size, compute0));
            checkCudaError(hipEventRecord(copyCEvent, compute0));
        }

        checkCudaError(hipStreamWaitEvent(compute0, copyXEvent, 0));
        add<<<grid, block, 0, compute0>>>(d_A, d_B, d_X, N);  // X_1 = A_0 + B_0
        checkCudaError(hipGetLastError());

        if (overlap) {
            checkCudaError(hipEventRecord(compXEvent, compute0));
        } else {
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_X, 0, size, compute0));
            checkCudaError(hipEventRecord(copyXEvent, compute0));
        }

        if (overlap) {
            checkCudaError(hipStreamWaitEvent(copyStream, compCEvent));
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_C, 0, size, copyStream));
            checkCudaError(hipEventRecord(copyCEvent, copyStream));

            checkCudaError(hipStreamWaitEvent(copyStream, compXEvent));
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_X, 0, size, copyStream));
            checkCudaError(hipEventRecord(copyXEvent, copyStream));
        }
        checkCudaError(hipSetDevice(1));

        checkCudaError(hipStreamWaitEvent(compute1, copyCEvent, 0));
        checkCudaError(hipStreamWaitEvent(compute1, compFEvent, 0));
        add<<<grid, block, 0, compute1>>>(d_D, d_E, d_F, N);
        checkCudaError(hipGetLastError());
        checkCudaError(hipEventRecord(compFEvent, compute1));

        checkCudaError(hipStreamWaitEvent(compute1, copyXEvent, 0));
        checkCudaError(hipStreamWaitEvent(compute1, compYEvent, 0));
        add<<<grid, block, 0, compute1>>>(d_D, d_E, d_Y, N);
        checkCudaError(hipGetLastError());
        checkCudaError(hipEventRecord(compYEvent, compute1));
    }

    checkCudaError(hipEventRecord(endEvent, 0));
    checkCudaError(hipEventSynchronize(endEvent));

    const auto chronoEnd = std::chrono::high_resolution_clock::now();
    const float chronoElapse = std::chrono::duration<float>(chronoEnd - chronoStart).count();
    std::cout << "chrono time: " << chronoElapse << std::endl;

    float elapse;
    checkCudaError(hipEventElapsedTime(&elapse, startEvent, endEvent));

    std::cout << "Total time (" << (overlap ? "Overlapping" : "Synchronous") << "): " << elapse << " ms\n";

    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_F, d_F, size, hipMemcpyDeviceToHost));
    // checkCudaError(cudaStreamSynchronize(compute0));
    // checkCudaError(cudaStreamSynchronize(compute1));

    maxError(h_C, iterations, N, true);
    maxError(h_F, iterations, N, false);

    checkCudaError(hipSetDevice(0));
    checkCudaError(hipFree(d_A));
    checkCudaError(hipFree(d_B));
    checkCudaError(hipFree(d_C));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipFree(d_D));
    checkCudaError(hipFree(d_E));
    checkCudaError(hipFree(d_F));

    checkCudaError(hipEventDestroy(startEvent));
    checkCudaError(hipEventDestroy(endEvent));
    checkCudaError(hipEventDestroy(compCEvent));
    checkCudaError(hipEventDestroy(compXEvent));
    checkCudaError(hipEventDestroy(compFEvent));
    checkCudaError(hipEventDestroy(compYEvent));
    checkCudaError(hipEventDestroy(copyCEvent));
    checkCudaError(hipEventDestroy(copyXEvent));

    checkCudaError(hipStreamDestroy(compute0));
    checkCudaError(hipStreamDestroy(compute1));

    checkCudaError(hipHostFree(h_A));
    checkCudaError(hipHostFree(h_B));
    checkCudaError(hipHostFree(h_C));
    checkCudaError(hipHostFree(h_D));
    checkCudaError(hipHostFree(h_F));

    return elapse;
}

int main() {
    // const int experiments = 8;
    // const int iterations = 100;
    // int base = 1;

    const int experiments = 1;
    const int iterations = 10;
    int base = 100000;

    for (int i = 0; i < experiments; i++) {
        const int N = base;
        base *= 10;
        std::cout << "=============================="
                << std::endl << "N = " << N << std::endl;
        std::cout << "Running synchronous version...\n";
        float sync = experiment(false, N, iterations);

        std::cout << "Running overlapping version...\n";
        float async = experiment(true, N, iterations);

        float speedup = (sync - async) / (sync);
        std::cout << "speedup: " << speedup << std::endl;
    }

    return 0;
}
