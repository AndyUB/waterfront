
#include <hip/hip_runtime.h>
#include <iostream>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void add(float* A, float* B, float* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        for (int i = 0; i < 32; i++) {
            float sinA = sinf(A[tid]);
            float cosA = cosf(A[tid]);
            float idA = sqrtf(sinA * sinA + cosA * cosA) * A[tid];
            float sinB = sinf(B[tid]);
            float cosB = cosf(B[tid]);
            float idB = sqrtf(sinB * sinB + cosB * cosB) * B[tid];
            float added = idA + idB;
            float sinAdded = sinf(added);
            float cosAdded = cosf(added);
            float idAdded = sqrtf(sinAdded * sinAdded + cosAdded * cosAdded) * added;
            C[tid] = idAdded;
        }
    }
}

void maxError(float *output, int iterations, int N, bool isC) {
    float maxErr = 0;
    int maxI = -1;
    const float epsilon = 1e-6;
    // basic comp:
    float expected = isC ? 3.0f : 6.0f;
    // heavy comp:
    // float expected = 4 * iterations + 3;
    // heavier comp:
    // float expected = 8 * iterations + 3;
    // if (!isC) expected += 3;
    for (int i = 0; i < N; ++i) {
        float diff = std::abs(output[i] - expected);
        if (diff > maxErr) {
            maxErr = diff;
            maxI = i;
        }
        if (diff > epsilon) {
            std::cout << "Error: " << (isC ? "C" : "F") << ": " << i << " (" << output[i] << ")!" << std::endl;
        }
    }
    std::cout << "Max error: " << maxErr << std::endl;
    if (maxI != -1) {
        std::cout << "at " << maxI << ": " << output[maxI] << std::endl;
    }
}

float experiment(bool overlap, int N, int iterations) {
    size_t size = N * sizeof(float);
    int block = 256;
    int grid = (N + block - 1) / block;

    float *d_A, *d_B, *d_C;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipMalloc((void**)&d_A, size));
    checkCudaError(hipMalloc((void**)&d_B, size));
    checkCudaError(hipMalloc((void**)&d_C, size));

    float *d_D, *d_E, *d_F;
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMalloc((void**)&d_D, size));
    checkCudaError(hipMalloc((void**)&d_E, size));
    checkCudaError(hipMalloc((void**)&d_F, size));

    float *h_A, *h_B, *h_C, *h_D, *h_F;
    checkCudaError(hipHostMalloc((void**)&h_A, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_B, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_C, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_D, size, hipHostMallocDefault));
    checkCudaError(hipHostMalloc((void**)&h_F, size, hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
        h_C[i] = -100.0f;
        h_D[i] = 3.0f;
    }

    hipEvent_t startEvent, endEvent, copyEvent, firstCompEvent;
    checkCudaError(hipEventCreate(&startEvent));
    checkCudaError(hipEventCreate(&endEvent));

    hipStream_t compute0, compute1, copyStream;
    checkCudaError(hipSetDevice(0));
    checkCudaError(hipStreamCreate(&compute0));
    checkCudaError(hipEventCreate(&firstCompEvent));
    checkCudaError(hipStreamCreate(&copyStream));
    checkCudaError(hipEventCreate(&copyEvent));
    checkCudaError(hipSetDevice(1));
    checkCudaError(hipStreamCreate(&compute1));

    checkCudaError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice));

    checkCudaError(hipEventRecord(startEvent, 0));

    for (int i = 0; i < iterations; ++i) {
        checkCudaError(hipSetDevice(0));
        add<<<grid, block, 0, compute0>>>(d_A, d_B, d_C, N);  // C_1 = A_0 + B_0
        checkCudaError(hipGetLastError());
        if (overlap) {
            checkCudaError(hipEventRecord(firstCompEvent, compute0));
            checkCudaError(hipStreamWaitEvent(copyStream, firstCompEvent));
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_C, 0, size, copyStream));
            checkCudaError(hipEventRecord(copyEvent, copyStream));
        } else {
            checkCudaError(hipMemcpyPeerAsync(d_E, 1, d_C, 0, size, compute0));
            checkCudaError(hipEventRecord(copyEvent, compute0));
        }
        checkCudaError(hipSetDevice(1));
        checkCudaError(hipStreamWaitEvent(compute1, copyEvent, 0));

        add<<<grid, block, 0, compute1>>>(d_D, d_E, d_F, N);
        checkCudaError(hipGetLastError());
    }

    checkCudaError(hipEventRecord(endEvent, 0));
    checkCudaError(hipEventSynchronize(endEvent));

    float elapse;
    checkCudaError(hipEventElapsedTime(&elapse, startEvent, endEvent));

    std::cout << "Total time (" << (overlap ? "Overlapping" : "Synchronous") << "): " << elapse << " ms\n";

    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_F, d_F, size, hipMemcpyDeviceToHost));
    // checkCudaError(cudaStreamSynchronize(compute0));
    // checkCudaError(cudaStreamSynchronize(compute1));

    maxError(h_C, iterations, N, true);
    maxError(h_F, iterations, N, false);

    checkCudaError(hipSetDevice(0));
    checkCudaError(hipFree(d_A));
    checkCudaError(hipFree(d_B));
    checkCudaError(hipFree(d_C));

    checkCudaError(hipSetDevice(1));
    checkCudaError(hipFree(d_D));
    checkCudaError(hipFree(d_E));
    checkCudaError(hipFree(d_F));

    checkCudaError(hipEventDestroy(startEvent));
    checkCudaError(hipEventDestroy(endEvent));
    checkCudaError(hipEventDestroy(copyEvent));

    checkCudaError(hipStreamDestroy(compute0));
    checkCudaError(hipStreamDestroy(compute1));

    checkCudaError(hipHostFree(h_A));
    checkCudaError(hipHostFree(h_B));
    checkCudaError(hipHostFree(h_C));
    checkCudaError(hipHostFree(h_D));
    checkCudaError(hipHostFree(h_F));

    return elapse;
}

int main() {
    const int experiments = 8;
    const int iterations = 100;
    int base = 1;

    for (int i = 0; i < experiments; i++) {
        const int N = base;
        base *= 10;
        std::cout << "=============================="
                << std::endl << "N = " << N << std::endl;
        std::cout << "Running synchronous version...\n";
        float sync = experiment(false, N, iterations);

        std::cout << "Running overlapping version...\n";
        float async = experiment(true, N, iterations);

        float speedup = (sync - async) / (sync);
        std::cout << "speedup: " << speedup << std::endl;
    }

    return 0;
}
